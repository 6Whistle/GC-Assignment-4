
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
using namespace std;

__global__ void addKernel(int *dev_c, const int *dev_a, const int *dev_b, const int WIDTH){
    int a_idx = (blockIdx.y * blockDim.y + threadIdx.y) * WIDTH;
    int b_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int c_idx = a_idx + b_idx;
    int sum = 0;

    for(int i = 0; i < WIDTH; i++, a_idx++, b_idx += WIDTH)
        sum += dev_a[a_idx] * dev_b[b_idx]; 
    dev_c[c_idx] = sum;

    return;
}

int main(void){
    const int WIDTH = 16;
    const int TILE_WIDTH = 2;
    int a[WIDTH][WIDTH], b[WIDTH][WIDTH], c[WIDTH][WIDTH];
    int c_check[WIDTH][WIDTH];
    int *dev_a, *dev_b, *dev_c;

    srand((unsigned int)time(NULL));

    for(int i = 0; i < WIDTH; i++){
        for(int j = 0; j < WIDTH; j++){
            a[i][j] = rand() % 10;
            b[i][j] = rand() % 10;
        }
    }

    hipMalloc((void **)&dev_a, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void **)&dev_b, WIDTH * WIDTH * sizeof(int));
    hipMalloc((void **)&dev_c, WIDTH * WIDTH * sizeof(int));

    hipMemcpy((void *)dev_a, (void *)a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy((void *)dev_b, (void *)b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice);

    dim3 DimGrid(WIDTH / TILE_WIDTH, WIDTH / TILE_WIDTH, 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    addKernel<<<DimGrid, DimBlock>>> (dev_c, dev_a, dev_b, WIDTH);

    hipDeviceSynchronize();
    hipMemcpy((void *)c, (void *)dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Matrix Multiplication" << endl;
    for(int i = 0; i < WIDTH; i++){
        for(int j = 0; j < WIDTH; j++)  cout << setw(4) << a[i][j];
        (i == WIDTH / 2) ? cout << "  *" : cout << "   ";

        for(int j = 0; j < WIDTH; j++)  cout << setw(4) << b[i][j];
        (i == WIDTH / 2) ? cout << "  =" : cout << "   ";

        cout << endl;
    }

    for(int i = 0; i < 4; i++){
        cout << i + 1 << " block : " << endl;
        int block_x = i / 2;
        int block_y = i % 2;
        int block_size = WIDTH / 2;
        for(int j = 0; j < block_size; j++){
            for(int k = 0; k < WIDTH / 2; k++)  cout << setw(4) << c[block_x * block_size + j][block_y * block_size + k];
            cout << endl;
        }
    }

    bool state = [&](){
        for(int i = 0; i < WIDTH; i++){
            for(int j = 0; j < WIDTH; j++){
                c_check[i][j] = 0;
                for(int k = 0; k < WIDTH; k++)  c_check[i][j] += a[i][k] * b[k][j];
                if(c[i][j] != c_check[i][j])    return false;
            }
        }
        return true;
    }();

    cout << "\n\nCheck Muliplication : " << (state == true ? "true" : "false") << endl;
    // for(int i = 0; i < WIDTH; i++){
    //     for(int j = 0; j < WIDTH; j++)  cout << setw(4) << c_check[i][j];
    //     cout << endl;
    // }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}